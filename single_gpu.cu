


#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>
#include <sys/time.h>

#define DIES 0
#define ALIVE 1
//#define DEBUG2 1

/* function to measure time taken */
double gettime(void) 
{
    struct timeval tval;

    gettimeofday(&tval, NULL);

    return( (double)tval.tv_sec + (double)tval.tv_usec/1000000.0 );
}

void printarray(int *a, int M, int N, FILE *fp) 
{
    int i, j;
    for (i = 0; i < M+2; i++) 
    {
        for (j = 0; j< N+2; j++)
            fprintf(fp, "%d ", a[i*(N+2) + j]);
        fprintf(fp, "\n");
    }
}

int check_array(int *a, int M, int N) 
{
    int value=0;
    for (int i = 1; i < M+1; i++)
        for (int j = 1; j < N+1; j++)
            value += a[i*(N+2) + j];
    return value;
}

/* cuda kerel to compute a step in the game */
__global__ void compute_kernel(int *life, int *temp, int N) 
{

    extern __shared__ int life_shared[];

    int i = blockIdx.x + 1;
    int j = threadIdx.x + 1;

    /* copy three rows of life matrix into shared memory */
    for (int k = j; k < N+1; k += blockDim.x) 
    {
        life_shared[k] = life[(i-1)*(N+2) + k];
        life_shared[N+2 + k] = life[(i)*(N+2) + k];
        life_shared[2*(N+2) + k] = life[(i+1)*(N+2) + k];


    }

    /* copy border values to shared memory*/
    if (threadIdx.x == 0) 
    {
        life_shared[0] = life[(i-1)*(N+2)];
        life_shared[N+2] = life[(i)*(N+2)];
        life_shared[2*(N+2)] = life[(i+1)*(N+2)];

        life_shared[N+1] = life[(i-1)*(N+2) + N+1];
        life_shared[N+2 + N+1] = life[(i)*(N+2) + N+1];
        life_shared[2*(N+2) + N+1] = life[(i+1)*(N+2) + N+1];
    }

    __syncthreads();

    for (int k = j; k < N+1; k += blockDim.x) 
    {
        /* find out the value of the current cell */
        int value = life_shared[(k-1)] + life_shared[k] + life_shared[(k+1)] + 
                    life_shared[(N+2) + (k-1)] + life_shared[(N+2) + (k+1)] + 
                    life_shared[2*(N+2) + (k-1)] + life_shared[2*(N+2) + k] + life_shared[2*(N+2) + (k+1)] ;
        
        /* check if the cell dies or life is born */
        if (life_shared[(N+2) + k]) 
        { // cell was alive in the earlier iteration
            if (value < 2 || value > 3) 
            {
                temp[i*(N+2) + k] = DIES ;
            }
            else // value must be 2 or 3, so no need to check explicitly
                temp[i*(N+2) + k] = ALIVE ; // no change
        } 
        else 
        { // cell was dead in the earlier iteration
            if (value == 3) 
            {
                temp[i*(N+2) + k] = ALIVE;
            }
            else
                temp[i*(N+2) + k] = DIES; // no change
        }
    }
}

void compute(int *life, int *temp, int M, int N) 
{

    /* set number of threads and number of blocks for cuda kernel */
    int numThreads = 1024;
    int numBlocks =N;
 
    /* set the dyamic amount of shared memory to be reserved per block*/
    unsigned sharedMemSize = (3 * (N+2) * sizeof(int));

    compute_kernel <<<numBlocks, numThreads, sharedMemSize>>> (life, temp, N);
}

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess)
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}
 
int main(int argc, char **argv) 
{
    int N, NTIMES, *life=NULL, *temp=NULL;
    int i, j, k;
    double t1, t2;

#if defined(DEBUG1) || defined(DEBUG2)
    FILE *fp;
    char GOL[32];
#endif
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    N = 10000;
    NTIMES = 5000;

    /* Allocate memory for both arrays */
    life = (int*) malloc((N+2)*(N+2)*sizeof(int));
    temp = (int*) malloc((N+2)*(N+2)*sizeof(int));

    /* Initialize the boundaries of the life matrix */
    for (i = 0; i < N+2; i++) 
    {
        life[i*(N+2)] = life[i*(N+2) + (N+1)] = DIES ;
        temp[i*(N+2)] = temp[i*(N+2) + (N+1)] = DIES ;
    }
    for (j = 0; j < N+2; j++) 
    {
        life[j] = life[(N+1)*(N+2) + j] = DIES ;
        temp[j] = temp[(N+1)*(N+2) + j] = DIES ;
    }

    /* Initialize the life matrix */
    for (i = 1; i < N+1; i++) 
    {
        for (j = 1; j< N+1; j++) 
        {
            if (drand48() < 0.5) 
	            life[i*(N+2) + j] = ALIVE ;
            else
	            life[i*(N+2) + j] = DIES ;
        }
    }

#ifdef DEBUG1
    /* Display the initialized life matrix */
    printf("Printing to file: output.%d.0\n",N);
    sprintf(GOL,"output.%d.0",N);
    fp = fopen("GOL.txt", "w");
    printarray(life, N, N, fp);
    fclose(fp);
#endif

    /* Define device memory pointers */
    int *life_device, *temp_device;

    /* Allocate global memory in device */
    hipMalloc(&life_device, (N+2)*(N+2)*sizeof(int));
    hipMalloc(&temp_device, (N+2)*(N+2)*sizeof(int));

    /* Initialize the device pointers */
    hipMemcpy(life_device, life, (N+2)*(N+2)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(temp_device, temp, (N+2)*(N+2)*sizeof(int), hipMemcpyHostToDevice);

    t1 = gettime();
    hipEventRecord(start);

    /* Play the game of life for given number of iterations */
    for (k = 0; k < NTIMES; k += 2) 
    {
        compute(life_device, temp_device, N, N);
        compute(temp_device, life_device, N, N);
    }
    gpuErrchk(hipDeviceSynchronize());
    hipEventRecord(stop);
    hipEventSynchronize(stop);
 
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Time Taken by Kernal: %6f milisec \n",milliseconds);
    t2 = gettime();

    /* Copy the result back to host */
    hipMemcpy(life, life_device, (N+2)*(N+2)*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(temp, temp_device, (N+2)*(N+2)*sizeof(int), hipMemcpyDeviceToHost);

    int life_remaining = check_array(life, N, N);
    printf("Total Time taken for size = %d after %d iterations = %f sec\n", N, k, t2-t1);
    printf("No. of cells alive after %d iterations = %d\n", k, life_remaining);

#ifdef DEBUG2
    /* Display the life matrix after k iterations */
    printf("Printing to file: output.%d.%d\n",N,k);
    sprintf(GOL,"output.%d.%d",N,k);
    fp = fopen("GOL.txt", "w");
    printarray(life, N, N, fp);
    fclose(fp);
#endif

    hipFree(temp_device);
    hipFree(life_device);
    free(life);
    free(temp);

    return 0;
}
